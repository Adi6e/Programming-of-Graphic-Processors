#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)  									\
do {											\
	hipError_t res = call;							\
	if (res != hipSuccess) {							\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								\
	}										\
} while(0)


texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int w, int h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	uchar4 p;
	for(y = idy; y < h; y += offsety)
		for(x = idx; x < w; x += offsetx) {
			p = tex2D(tex, x, y);
			out[(h - 1 - y) * w + x] = make_uchar4(~p.x, ~p.y, ~p.z, p.w);
		}
}

int main() {
	int w, h;
	FILE *fp = fopen("in.data", "rb");
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));

	CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

	tex.normalized = false;
	tex.filterMode = hipFilterModePoint;	
	tex.channelDesc = ch;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;

	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

	kernel<<< dim3(16, 16), dim3(32, 32) >>>(dev_out, w, h);
	CSC(hipGetLastError());
	
	CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
	CSC(hipUnbindTexture(tex));
	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	fp = fopen("out.data", "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	return 0;
}
