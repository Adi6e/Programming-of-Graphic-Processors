#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define CSC(call)  									\
do {											\
	hipError_t res = call;							\
	if (res != hipSuccess) {							\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								\
	}										\
} while(0)

#define MIN(a,b) ((a) < (b) ? (a) : (b))

texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int w, int h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	uchar4 p1, p2, p3, p4;
	for(int y = idy; y < h; y += offsety){
		for(int x = idx; x < w; x += offsetx) {
			p1 = tex2D(tex, x, y);
			p2 = tex2D(tex, x + 1, y + 1);
			p3 = tex2D(tex, x + 1, y);
			p4 = tex2D(tex, x, y + 1);
			float Y1 = 0.299 * p1.x + 0.587 * p1.y + 0.114 * p1.z;
			float Y2 = 0.299 * p2.x + 0.587 * p2.y + 0.114 * p2.z;
			float Y3 = 0.299 * p3.x + 0.587 * p3.y + 0.114 * p3.z;
			float Y4 = 0.299 * p4.x + 0.587 * p4.y + 0.114 * p4.z;
			float gx = Y2 - Y1;
			float gy = Y4 - Y3;
			int g = MIN(255, sqrt(gx * gx + gy * gy));
			out[y * w + x] = make_uchar4(g, g, g, p1.w);
		}
	}
}

int main() {
	int w, h;
	char in[255], out[255];
	scanf("%s", in);
	scanf("%s", out);
	FILE *fp = fopen(in, "rb");
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));

	CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

	tex.normalized = false;
	tex.filterMode = hipFilterModePoint;	
	tex.channelDesc = ch;
	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;

	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

	hipEvent_t start, stop;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start));

	kernel<<< dim3(32, 32), dim3(32, 32)>>> (dev_out, w, h);
    CSC(hipDeviceSynchronize());
	CSC(hipGetLastError());
	
	CSC(hipEventRecord(stop));
	CSC(hipEventSynchronize(stop));
	float t;
	CSC(hipEventElapsedTime(&t, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));

	printf("time = %f ms\n", t);

	CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
	CSC(hipUnbindTexture(tex));
	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	fp = fopen(out, "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	return 0;
}
