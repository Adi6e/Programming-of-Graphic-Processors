
#include <hip/hip_runtime.h>
#include <stdio.h>

// nvprof -e divergent_branch,global_store_transaction,l1_shared_bank_conflict,l1_local_load_hit ./a.out
//
// divergent_branch - дивергенция нитей
// global_store_transaction - кол-во транзакций к глобальной памяти
// l1_shared_bank_conflict - кол-во конфликтов банков памяти при работе с разделяемой памятью
// l1_local_load_hit - перенос переменных из регистровой памяти в локальную

// sm_efficiency - загрузка мультипроцессоров

#define _index(i) ((i) + ((i) >> 5))

__global__ void kernel_shared(float *src, float *dst, int n) {
	__shared__ float buff[_index(32 * 32)];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < n && idy < n)
		buff[_index(32 * threadIdx.x + threadIdx.y)] = src[idy * n + idx];	
	__syncthreads();	
	idx = blockIdx.x * blockDim.x + threadIdx.y;
	idy = blockIdx.y * blockDim.y + threadIdx.x;
	if (idx < n && idy < n)	
		dst[idx * n + idy] = buff[_index(32 * threadIdx.y + threadIdx.x)];
}

__global__ void kernel_shared1(float *src, float *dst, int n) {
	__shared__ float buff[32][33];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < n && idy < n)
		buff[threadIdx.x][threadIdx.y] = src[idy * n + idx];	
	__syncthreads();	
	idx = blockIdx.x * blockDim.x + threadIdx.y;
	idy = blockIdx.y * blockDim.y + threadIdx.x;
	if (idx < n && idy < n)	
		dst[idx * n + idy] = buff[threadIdx.y][threadIdx.x];
}

__global__ void kernel(float *src, float *dst, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx < n && idy < n)
		dst[idx * n + idy] = src[idy * n + idx]; 
}

int main() {
	int i, j, n = 1000;
	float *src = (float *)malloc(sizeof(float) * n * n);
	float *dst = (float *)malloc(sizeof(float) * n * n);
	for(i = 0; i < n * n; i++)
		src[i] = i;
	float *dev_src, *dev_dst;
	hipMalloc(&dev_src, sizeof(float) * n * n);
	hipMalloc(&dev_dst, sizeof(float) * n * n);
	hipMemcpy(dev_src, src, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemset(dev_dst, 0,  sizeof(float) * n * n);

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	kernel<<<dim3(32, 32), dim3(32, 32)>>>(dev_src, dev_dst, n);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	fprintf(stderr, "time = %f\n", time);

	hipMemcpy(dst, dev_dst, sizeof(float) * n * n, hipMemcpyDeviceToHost); 

	for(i = 0; i < n; i++)
		for(j = 0; j < n; j++)	
			if (src[j * n + i] != dst[i * n + j])
				fprintf(stderr, "ERROR!!!\n");

	hipFree(dev_src);
	hipFree(dev_dst);
	free(src);
	free(dst);
	return 0;
}
