
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define sqr(x) ((x)*(x))

#define CSC(call) 					\
do { 							\
	hipError_t status = call;			\
	if (status != hipSuccess) {							\
		fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));		\
		exit(0);								\
	}										\
} while(0)

__global__ void kernel(float *arr, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	while (idx < n) {
		assert(idx < n);
		arr[idx] = __sinf(sqr(arr[idx]));
		idx += offset;
	}
}

int main() {
	long int i, n = 100000000;
	float *arr = (float *)malloc(sizeof(float) * n);
	for(i = 0; i < n; i++)
		arr[i] = i / (float)(n - 1);
	float *dev_arr;
	
	CSC(hipMalloc(&dev_arr, sizeof(float) * n));
	CSC(hipMemcpy(dev_arr, arr, sizeof(float) * n, hipMemcpyHostToDevice));

	hipEvent_t start, stop;
	CSC(hipEventCreate(&start));
	CSC(hipEventCreate(&stop));
	CSC(hipEventRecord(start));
	
	kernel<<<256, 256>>>(dev_arr, n);
	CSC(hipDeviceSynchronize());
	CSC(hipGetLastError());

	CSC(hipEventRecord(stop));
	CSC(hipEventSynchronize(stop));
	float t;
	CSC(hipEventElapsedTime(&t, start, stop));
	CSC(hipEventDestroy(start));
	CSC(hipEventDestroy(stop));

	printf("time = %f ms\n", t);

	CSC(hipMemcpy(arr, dev_arr, sizeof(float) * n, hipMemcpyDeviceToHost));
	for(i = n - 10; i < n; i++)
		printf("%f ", arr[i]);
	printf("\n");

	CSC(hipFree(dev_arr));
	free(arr);
	return 0;
}
