
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define CSC(call) 					\
do { 							\
	hipError_t status = call;			\
	if (status != hipSuccess) {							\
		fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status));		\
		exit(0);								\
	}										\
} while(0)

__global__ void kernel(float *first_vec, float *second_vec, float *res_vec, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	while (idx < n) 
    {
        if (first_vec[idx] > second_vec[idx])
            res_vec[idx] = first_vec[idx];
        else
            res_vec[idx] = second_vec[idx];
        idx += offset;
	}
}

int main() 
{
	int i, n;
    scanf("%d", &n);

	float *first_arr = (float *)malloc(sizeof(float) * n);
	float *second_arr = (float *)malloc(sizeof(float) * n);
    float *res_arr = (float *)malloc(sizeof(float) * n);

	for(i = 0; i < n; i++)
		scanf("%f", &first_arr[i]);
	for(i = 0; i < n; i++)
		scanf("%f", &second_arr[i]);
	for(i = 0; i < n; i++)
		res_arr[i] = i;

	float *dev_arr1, *dev_arr2, *dev_arr3;
	CSC(hipMalloc(&dev_arr1, sizeof(float) * n));
	CSC(hipMemcpy(dev_arr1, first_arr, sizeof(float) * n, hipMemcpyHostToDevice));
	CSC(hipMalloc(&dev_arr2, sizeof(float) * n));
	CSC(hipMemcpy(dev_arr2, second_arr, sizeof(float) * n, hipMemcpyHostToDevice));
	CSC(hipMalloc(&dev_arr3, sizeof(float) * n));
	CSC(hipMemcpy(dev_arr3, res_arr, sizeof(float) * n, hipMemcpyHostToDevice));

	kernel<<<32, 32>>>(dev_arr1, dev_arr2, dev_arr3, n);
	CSC(hipDeviceSynchronize());
	CSC(hipGetLastError());

	CSC(hipMemcpy(res_arr, dev_arr3, sizeof(float) * n, hipMemcpyDeviceToHost));

	for(i = 0; i < n; i++)
		printf("%.10e ", res_arr[i]);
	printf("\n");

	CSC(hipFree(dev_arr1));
    CSC(hipFree(dev_arr2));
    CSC(hipFree(dev_arr3));
	free(first_arr);
    free(second_arr);
    free(res_arr);
	return 0;
}
